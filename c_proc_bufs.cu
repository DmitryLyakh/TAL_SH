#include "hip/hip_runtime.h"
/** Explicit memory management for the accelerator-enabled
implementation of the tensor algebra library TAL-SH:
CP-TAL (TAL for CPU), NV-TAL (TAL for NVidia GPU),
XP-TAL (TAL for Intel Xeon Phi), AM-TAL (TAL for AMD GPU).
REVISION: 2015/07/20
Copyright (C) 2015 Dmitry I. Lyakh (email: quant4me@gmail.com)
Copyright (C) 2015 Oak Ridge National Laboratory (UT-Battelle)

This source file is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
-------------------------------------------------------------------------------
OPTIONS:
 # -DNO_GPU: disables GPU usage (CPU only memory management).
**/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "tensor_algebra.h"

#define GPU_MEM_PART_USED 90         //percentage of free GPU global memory to be actually allocated for GPU argument buffers
#define MEM_ALIGN GPU_CACHE_LINE_LEN //memory alignment (in bytes) for argument buffers
#define BLCK_BUF_DEPTH_HOST 4        //number of distinct tensor block buffer levels on Host
#define BLCK_BUF_TOP_HOST 3          //number of argument buffer entries of the largest size (level 0) on Host: multiple of 3
#define BLCK_BUF_BRANCH_HOST 3       //branching factor for each subsequent buffer level on Host
#define BLCK_BUF_DEPTH_GPU 3         //number of distinct tensor block buffer levels on GPU
#define BLCK_BUF_TOP_GPU 3           //number of argument buffer entries of the largest size (level 0) on GPU: multiple of 3
#define BLCK_BUF_BRANCH_GPU 2        //branching factor for each subsequent buffer level on GPU

static int VERBOSE=1; //verbosity (for errors)
static int DEBUG=0;   //debugging

//DERIVED TYPES:
// Argument buffer configuration:
typedef struct{
 int buf_top;    //amount of top-level blocks (of the largest size)
 int buf_depth;  //number of levels
 int buf_branch; //branching factor for each subsequent level
} ab_conf_t;

//MODULE DATA:
static void *arg_buf_host; //base address of the argument buffer in Host memory (page-locked)
static void *arg_buf_gpu[MAX_GPUS_PER_NODE]; //base addresses of argument buffers in GPUs Global memories
static size_t arg_buf_host_size=0; //total size of the Host argument buffer in bytes
static size_t arg_buf_gpu_size[MAX_GPUS_PER_NODE]; //total sizes of each GPU argument buffer in bytes
static int max_args_host=0; //max number of arguments (those of the lowest size level) which can reside in Host buffer
static int max_args_gpu[MAX_GPUS_PER_NODE]; //max number of arguments (those of the lowest size level) which can reside in a GPU buffer: will be overtaken by MAX_GPU_ARGS
static size_t blck_sizes_host[BLCK_BUF_DEPTH_HOST]; //distinct tensor block buffered sizes (in bytes) on Host
static size_t blck_sizes_gpu[MAX_GPUS_PER_NODE][BLCK_BUF_DEPTH_GPU]; //distinct tensor block buffered sizes (in bytes) on GPUs
static int const_args_link[MAX_GPUS_PER_NODE][MAX_GPU_ARGS]; //linked list of free entries in constant memory banks for each GPU
static int const_args_ffe[MAX_GPUS_PER_NODE]; //FFE of the const_args_link[] for each GPU
static size_t *abh_occ=NULL; //occupation status for each buffer entry in Host argument buffer (*arg_buf_host)
static size_t *abg_occ[MAX_GPUS_PER_NODE]; //occupation status for each buffer entry in GPU argument buffers(*arg_buf_gpu)
static size_t abh_occ_size=0; //total number of entries in the multi-level Host argument buffer occupancy table
static size_t abg_occ_size[MAX_GPUS_PER_NODE]; //total numbers of entries in the multi-level GPUs argument buffer occupancy tables

//LOCAL (PRIVATE) FUNCTION PROTOTYPES:
static int const_args_link_init(int gpu_beg, int gpu_end);
static int ab_get_2d_pos(ab_conf_t ab_conf, int entry_num, int *level, int *offset);
static int ab_get_1d_pos(ab_conf_t ab_conf, int level, int offset);
static int ab_get_parent(ab_conf_t ab_conf, int level, int offset);
static int ab_get_1st_child(ab_conf_t ab_conf, int level, int offset);
static size_t ab_get_offset(ab_conf_t ab_conf, int level, int offset, const size_t *blck_sizes);
static int get_buf_entry(ab_conf_t ab_conf, size_t bsize, void *arg_buf_ptr, size_t *ab_occ, size_t ab_occ_size,
                         const size_t *blck_sizes, char **entry_ptr, int *entry_num);
static int free_buf_entry(ab_conf_t ab_conf, size_t *ab_occ, size_t ab_occ_size, const size_t *blck_sizes, int entry_num);
//------------------------------------------------------------------------------------------------------------------------

//FUNCTION DEFINITIONS:
char* ptr_offset(char *byte_ptr, size_t byte_offset){char *addr=&byte_ptr[byte_offset]; return addr;}

static int ab_get_2d_pos(ab_conf_t ab_conf, int entry_num, int *level, int *offset)
/** Given an argument buffer entry number, this function returns the
corresponding buffer level and offset within that level **/
{
 int i,j,k,m;
 if(entry_num >= 0){
  m=ab_conf.buf_top; k=0; j=m;
  for(i=0;i<ab_conf.buf_depth;i++){
   if(entry_num<j){*level=i; *offset=entry_num-k; return 0;};
   m*=ab_conf.buf_branch; k=j; j=k+m;
  }
  return 1; //entry number is out of range
 }else{
  return 2;
 }
}

static int ab_get_1d_pos(ab_conf_t ab_conf, int level, int offset)
/** Given a buffer level and offset within it,
this function returns the plain buffer entry number **/
{
 int i,j,k;
 if(level >= 0 && level < ab_conf.buf_depth && offset >= 0){
  if(level == 0) return offset; j=ab_conf.buf_top; k=ab_conf.buf_top;
  for(i=1;i<ab_conf.buf_depth;i++){k*=ab_conf.buf_branch; if(i==level) break; j+=k;}
  if(offset < k){return j+offset;}else{return -1;}
 }else{
  return -2; //invalid buffer level
 }
}

static int ab_get_parent(ab_conf_t ab_conf, int level, int offset)
/** This function returns the offset of the parent of a given buffer entry {level, offset} **/
{
 if(level >= 0 && level < ab_conf.buf_depth && offset >= 0 && ab_conf.buf_branch > 0){
  return offset/ab_conf.buf_branch;
 }else{
  return -1;
 }
}

static int ab_get_1st_child(ab_conf_t ab_conf, int level, int offset)
{
/** This function returns the offset of the 1st child for a given buffer entry {level, offset} **/
 if(level >= 0 && level < ab_conf.buf_depth && offset >= 0 && ab_conf.buf_branch > 0){
  return offset*ab_conf.buf_branch;
 }else{
  return -1;
 }
}

static size_t ab_get_offset(ab_conf_t ab_conf, int level, int offset, const size_t *blck_sizes)
/** This function returns a byte offset in the argument buffer space
corresponding to a given buffer entry {level, offset}.
Note that the base address of the argument buffer must be added a posteriori!
No arguments bounds check here! **/
{
 int i,j;
 size_t ab_offset=0;
 ab_offset=offset*blck_sizes[level]; j=offset;
 for(i=level;i>0;i--){
  j=ab_get_parent(ab_conf,i,j);
  ab_offset+=(blck_sizes[i-1]%ab_conf.buf_branch)*j;
 }
 return ab_offset;
}

int arg_buf_allocate(size_t *arg_buf_size, int *arg_max, int gpu_beg, int gpu_end)
/** This function initializes all argument buffers on the Host and GPUs in the range [gpu_beg..gpu_end].
INPUT:
 # arg_buf_size - requested size of the page-locked Host argument buffer in bytes;
 # [gpu_beg..gpu_end] - range of GPUs assigned to the current MPI process;
OUTPUT:
 # arg_buf_size - actual size of the allocated page-locked Host argument buffer in bytes;
 # arg_max - max number of arguments the Host buffer can contain (those of the lowest size level).
**/
{
 size_t hsize,total,mem_alloc_dec;
 int i,j,err_code;
#ifndef NO_GPU
 hipError_t err=hipSuccess;
#endif

 *arg_max=0; abh_occ=NULL; abh_occ_size=0; max_args_host=0; arg_buf_host_size=0;
 for(i=0;i<MAX_GPUS_PER_NODE;i++){abg_occ[i]=NULL; abg_occ_size[i]=0; max_args_gpu[i]=0; arg_buf_gpu_size[i]=0;}
//Allocate the Host argument buffer:
 mem_alloc_dec=MEM_ALIGN*BLCK_BUF_TOP_HOST; for(i=1;i<BLCK_BUF_DEPTH_HOST;i++) mem_alloc_dec*=BLCK_BUF_BRANCH_HOST;
 hsize=*arg_buf_size; hsize-=hsize%mem_alloc_dec; err_code=1;
 while(hsize > mem_alloc_dec){
#ifndef NO_GPU
  err=hipHostAlloc(&arg_buf_host,hsize,hipHostMallocPortable);
  if(err != hipSuccess){
   hsize-=mem_alloc_dec;
  }else{
   *arg_buf_size=hsize; arg_buf_host_size=hsize; err_code=0;
   if(DEBUG) printf("\n#DEBUG(c_proc_bufs.cu:arg_buf_allocate): Pinned Host argument buffer address/size: %p %lld\n",arg_buf_host,(long long)hsize); //debug
   break;
  }
#else
  arg_buf_host=malloc(hsize);
  if(arg_buf_host == NULL){
   hsize-=mem_alloc_dec;
  }else{
   *arg_buf_size=hsize; arg_buf_host_size=hsize; err_code=0;
   if(DEBUG) printf("\n#DEBUG(c_proc_bufs.cu:arg_buf_allocate): Host buffer address/size: %p %lld\n",arg_buf_host,(long long)hsize); //debug
   break;
  }
#endif
 }
 if(err_code == 0){
//Set buffered block sizes hierarchy (buffer levels) for the Host argument buffer:
  hsize=BLCK_BUF_TOP_HOST; max_args_host=BLCK_BUF_TOP_HOST; blck_sizes_host[0]=arg_buf_host_size/BLCK_BUF_TOP_HOST;
  for(i=1;i<BLCK_BUF_DEPTH_HOST;i++){
   blck_sizes_host[i]=blck_sizes_host[i-1]/BLCK_BUF_BRANCH_HOST; max_args_host*=BLCK_BUF_BRANCH_HOST;
   hsize+=max_args_host;
  }
  *arg_max=max_args_host;
//Initialize the Host argument buffer occupancy table:
  abh_occ=(size_t*)malloc(hsize*sizeof(size_t)); if(abh_occ == NULL) return 1; //Host buffer occupancy table
  abh_occ_size=hsize;
  for(hsize=0;hsize<abh_occ_size;hsize++){abh_occ[hsize]=0;} //initialize zero occupancy for each buffer entry
#ifndef NO_GPU
//Allocate GPUs buffers, if needed:
  if(gpu_beg >= 0 && gpu_end >= gpu_beg){ //GPU exist for this MPI process
   err=hipGetDeviceCount(&i); if(err != hipSuccess) return 2;
   if(gpu_end < MAX_GPUS_PER_NODE && gpu_end < i){
    err_code=init_gpus(gpu_beg,gpu_end); if(err_code < 0) return 9;
// Constant memory banks for all GPUs:
    err_code=const_args_link_init(gpu_beg,gpu_end); if(err_code != 0) return 3;
// Global memory banks for each GPU:
    mem_alloc_dec=MEM_ALIGN*BLCK_BUF_TOP_GPU; for(i=1;i<BLCK_BUF_DEPTH_GPU;i++) mem_alloc_dec*=BLCK_BUF_BRANCH_GPU;
    for(i=gpu_beg;i<=gpu_end;i++){
     if(gpu_is_mine(i) != 0){ //Initialize only my GPUs
      err=hipSetDevice(i); if(err != hipSuccess) return 4;
      err=hipMemGetInfo(&hsize,&total); if(err != hipSuccess) return 5;
      hsize=(size_t)(float(hsize)/100.0f*float(GPU_MEM_PART_USED)); hsize-=hsize%mem_alloc_dec; err_code=1;
      while(hsize > mem_alloc_dec){
       err=hipMalloc(&arg_buf_gpu[i],hsize);
       if(err != hipSuccess){
        hsize-=mem_alloc_dec;
       }else{
        arg_buf_gpu_size[i]=hsize; err_code=0;
        if(DEBUG) printf("\n#DEBUG(c_proc_bufs.cu:arg_buf_allocate): GPU#%d argument buffer address/size: %p %lld\n",i,arg_buf_gpu[i],(long long)hsize); //debug
        break;
       }
      }
      if(err_code == 0){
// Set buffered block sizes hierarchy (buffer levels) for each GPU argument buffer:
       hsize=BLCK_BUF_TOP_GPU; max_args_gpu[i]=BLCK_BUF_TOP_GPU; blck_sizes_gpu[i][0]=arg_buf_gpu_size[i]/BLCK_BUF_TOP_GPU;
       for(j=1;j<BLCK_BUF_DEPTH_GPU;j++){
        blck_sizes_gpu[i][j]=blck_sizes_gpu[i][j-1]/BLCK_BUF_BRANCH_GPU; max_args_gpu[i]*=BLCK_BUF_BRANCH_GPU;
        hsize+=max_args_gpu[i];
       }
       if(max_args_gpu[i] > MAX_GPU_ARGS) return 6; //Increase MAX_GPU_ARGS and recompile
// Initialize each GPU argument buffer occupancy table:
       abg_occ[i]=(size_t*)malloc(hsize*sizeof(size_t)); if(abg_occ[i] == NULL) return 7; //GPU#i buffer occupancy table
       abg_occ_size[i]=hsize;
       for(hsize=0;hsize<abg_occ_size[i];hsize++){abg_occ[i][hsize]=0;} //initialize each buffer entry to zero occupancy
      }else{
       return 8;
      }
     }
    }
   }else{
    return 10;
   }
  }
#endif
 }else{
  return 11;
 }
 return 0;
}

int arg_buf_deallocate(int gpu_beg, int gpu_end)
/** This function deallocates all argument buffers on the Host and GPUs in the range [gpu_beg..gpu_end] **/
{
 int i,err_code;
#ifndef NO_GPU
 hipError_t err=hipSuccess;
#endif
 err_code=0;
 if(abh_occ != NULL) free(abh_occ); abh_occ=NULL; abh_occ_size=0; max_args_host=0;
 for(i=0;i<MAX_GPUS_PER_NODE;i++){
  if(abg_occ[i] != NULL) free(abg_occ[i]); abg_occ[i]=NULL; abg_occ_size[i]=0; max_args_gpu[i]=0;
 }
 arg_buf_host_size=0;
#ifndef NO_GPU
 err=hipHostFree(arg_buf_host);
 if(err != hipSuccess){
  if(VERBOSE) printf("\n#ERROR(c_proc_bufs.cu:arg_buf_deallocate): Host argument buffer deallocation failed!");
  err_code=1;
 }
 if(gpu_beg >= 0 && gpu_end >= gpu_beg){
  for(i=gpu_beg;i<=gpu_end;i++){
   if(i < MAX_GPUS_PER_NODE){
    if(gpu_is_mine(i) != 0){
     err=hipSetDevice(i); if(err == hipSuccess){
      arg_buf_gpu_size[i]=0;
      err=hipFree(arg_buf_gpu[i]);
      if(err != hipSuccess){
       if(VERBOSE) printf("\n#ERROR(c_proc_bufs.cu:arg_buf_deallocate): GPU# %d argument buffer deallocation failed!",i);
       err_code++;
      }
     }else{
      if(VERBOSE) printf("\n#ERROR(c_proc_bufs.cu:arg_buf_deallocate): Unable to set GPU# %d!",i);
      err_code++;
     }
    }
   }else{
    err_code++;
   }
  }
  i=free_gpus(gpu_beg,gpu_end); if(i != 0) err_code+=1000;
 }
#else
 free(arg_buf_host); arg_buf_host=NULL;
#endif
 return err_code;
}

int arg_buf_clean_host()
/** Returns zero if all entries of the Host argument buffer are free. **/
{
 for(size_t i=0;i<abh_occ_size;i++){if(abh_occ[i] != 0) return (int)(i+1);}
 return 0;
}

int arg_buf_clean_gpu(int gpu_num)
/** Returns zero if all entries of the GPU#gpu_num argument buffer are free. **/
{
 if(gpu_num >= 0 && gpu_num < MAX_GPUS_PER_NODE){
  for(size_t i=0;i<abg_occ_size[gpu_num];i++){if(abg_occ[gpu_num][i] != 0) return (int)(i+1);}
 }else{
  return -1;
 }
 return 0;
}

int get_blck_buf_sizes_host(size_t *blck_sizes)
/** This function returns the registered block (buffered) sizes for each level of the Host argument buffer **/
{
 for(int i=0;i<BLCK_BUF_DEPTH_HOST;i++){blck_sizes[i]=blck_sizes_host[i];}
 return BLCK_BUF_DEPTH_HOST;
}

int get_blck_buf_sizes_gpu(int gpu_num, size_t *blck_sizes)
/** This function returns the registered block (buffered) sizes for each level of the GPU#gpu_num argument buffer **/
{
 for(int i=0;i<BLCK_BUF_DEPTH_GPU;i++){blck_sizes[i]=blck_sizes_gpu[gpu_num][i];}
 return BLCK_BUF_DEPTH_GPU;
}

static int get_buf_entry(ab_conf_t ab_conf, size_t bsize, void *arg_buf_ptr, size_t *ab_occ, size_t ab_occ_size,
                         const size_t *blck_sizes, char **entry_ptr, int *entry_num)
/** This function finds an appropriate argument buffer entry in any given argument buffer **/
{
 int i,j,k,l,m,n;
// if(DEBUG) printf("\n#DEBUG(c_proc_bufs.cu:get_buf_entry): %lu %lu\n",bsize,blck_sizes[0]); //debug
 *entry_ptr=NULL; *entry_num=-1;
 n=0; j=0; i=0; l=0; //l is a base offset within level i
 while(i<ab_conf.buf_depth){ //argument buffer level
  if(i > 0){k=ab_conf.buf_branch;}else{k=ab_conf.buf_top;};
  j=l%k; l-=j; j+=n;
  while(j<k){ //(l+j) is an offset within level i
   m=ab_get_1d_pos(ab_conf,i,l+j); if(m < 0 || m >= ab_occ_size) return 1; //m is an absolute offset in an occupancy table
//   if(DEBUG) printf("\n#DEBUG(c_proc_bufs.cu:get_buf_entry): Current level/offset/sizes: %d %d %d \n",i,l+j,blck_sizes[i]); //debug
   if(bsize <= blck_sizes[i]-ab_occ[m]){ //there is a good chance to find a free entry along this path
    if(i == ab_conf.buf_depth-1 && ab_occ[m] == 0){
     *entry_num=m; *entry_ptr=&(((char*)arg_buf_ptr)[ab_get_offset(ab_conf,i,l+j,blck_sizes)]); //entry found
     break;
    }else{
     if(blck_sizes[i+1] < bsize && ab_occ[m] == 0){
      *entry_num=m; *entry_ptr=&(((char*)arg_buf_ptr)[ab_get_offset(ab_conf,i,l+j,blck_sizes)]); //entry found
      break;
     }else{
      if(i < ab_conf.buf_depth-1){if(blck_sizes[i+1] >= bsize) break;} //initiate passing to the next level
     }
    }
   }
   j++; //horizontal shift
  } //enddo j
  if(*entry_num >= 0) break; //entry found
  if(j < k){ //proceed to the next level
   l=ab_get_1st_child(ab_conf,i,l+j); if(l < 0 || l >= ab_occ_size) return 2; i++; n=0; //go to the next level
  }else{ //back to the upper level
   if(i > 0){
    l=ab_get_parent(ab_conf,i,l); if(l < 0 || l >= ab_occ_size) return 3; i--; n=1; //return to the previous level
   }else{
    break;
   }
  }
 } //enddo i
 if(*entry_num >= 0 && *entry_num < ab_occ_size){
  k=blck_sizes[i]; ab_occ[m]=k;
  while(i>0){ //modify occupancy of the upper-level parental entries
   l=ab_get_parent(ab_conf,i,l); i--; m=ab_get_1d_pos(ab_conf,i,l); if(m < 0 || m >= ab_occ_size) return 4;
   ab_occ[m]+=k;
  }
 }else{
  return 5; //no appropriate entry found: not an error
 }
 return 0;
}

static int free_buf_entry(ab_conf_t ab_conf, size_t *ab_occ, size_t ab_occ_size, const size_t *blck_sizes, int entry_num)
/** This function releases an argument buffer entry in any given argument buffer **/
{
 int i,j,k,m;
 k=ab_get_2d_pos(ab_conf,entry_num,&i,&j); if(k != 0) return 1;
 if(ab_occ[entry_num] == blck_sizes[i]){ //buffer entries are always occupied as a whole
  k=blck_sizes[i]; ab_occ[entry_num]=0;
  while(i>0){ //modify occupancy of the upper-level parental entries
   j=ab_get_parent(ab_conf,i,j); i--; m=ab_get_1d_pos(ab_conf,i,j); if(m < 0 || m >= ab_occ_size) return 2;
   ab_occ[m]-=k;
  }
 }else{
  return 3;
 }
 return 0;
}

int get_buf_entry_host(size_t bsize, char **entry_ptr, int *entry_num)
/** This function returns a pointer to a free argument buffer space in the Host argument buffer.
INPUT:
 # bsize - requested size of a tensor block (in bytes);
OUTPUT:
 # entry_ptr - pointer to a free space in the argument buffer where the tensor block or packet can be put;
 # entry_num - entry number corresponding to the free space assigned to the tensor block or packet.
**/
{
 int err_code=0;
 ab_conf_t ab_conf;
 ab_conf.buf_top=BLCK_BUF_TOP_HOST; ab_conf.buf_depth=BLCK_BUF_DEPTH_HOST; ab_conf.buf_branch=BLCK_BUF_BRANCH_HOST;
 err_code=get_buf_entry(ab_conf,bsize,arg_buf_host,abh_occ,abh_occ_size,blck_sizes_host,entry_ptr,entry_num);
// if(err_code == 0 && DEBUG != 0) printf("\n#DEBUG(c_proc_bufs.cu:get_buf_entry_host): Entry allocated: %d %p\n",*entry_num,*entry_ptr); //debug
 return err_code;
}

int free_buf_entry_host(int entry_num)
/** This function releases a Host argument buffer entry.
INPUT:
 # entry_num - argument buffer entry number.
**/
{
 int err_code=0;
 ab_conf_t ab_conf;
 ab_conf.buf_top=BLCK_BUF_TOP_HOST; ab_conf.buf_depth=BLCK_BUF_DEPTH_HOST; ab_conf.buf_branch=BLCK_BUF_BRANCH_HOST;
 err_code=free_buf_entry(ab_conf,abh_occ,abh_occ_size,blck_sizes_host,entry_num);
// if(err_code == 0 && DEBUG != 0) printf("\n#DEBUG(c_proc_bufs.cu:free_buf_entry_host): Entry deallocated: %d\n",entry_num); //debug
 return err_code;
}

int get_buf_entry_gpu(int gpu_num, size_t bsize, char **entry_ptr, int *entry_num)
/** This function returns a pointer to a free argument buffer space in the GPU#gpu_num argument buffer.
INPUT:
 # gpu_num - GPU number;
 # bsize - requested size of a tensor block (in bytes);
OUTPUT:
 # entry_ptr - pointer to a free space in the argument buffer where the tensor block elements can be put;
 # entry_num - entry number corresponding to the free space assigned to the tensor block elements.
**/
{
 int err_code=0;
 ab_conf_t ab_conf;
 ab_conf.buf_top=BLCK_BUF_TOP_GPU; ab_conf.buf_depth=BLCK_BUF_DEPTH_GPU; ab_conf.buf_branch=BLCK_BUF_BRANCH_GPU;
 err_code=get_buf_entry(ab_conf,bsize,arg_buf_gpu[gpu_num],abg_occ[gpu_num],abg_occ_size[gpu_num],&blck_sizes_gpu[gpu_num][0],entry_ptr,entry_num);
// if(err_code == 0 && DEBUG != 0) printf("\n#DEBUG(c_proc_bufs.cu:get_buf_entry_gpu): Entry allocated: %d %d %p\n",gpu_num,*entry_num,*entry_ptr); //debug
 return err_code;
}

int free_buf_entry_gpu(int gpu_num, int entry_num)
/** This function releases a GPU#gpu_num argument buffer entry.
INPUT:
 # gpu_num - GPU number;
 # entry_num - argument buffer entry number.
**/
{
 int err_code=0;
 ab_conf_t ab_conf;
 ab_conf.buf_top=BLCK_BUF_TOP_GPU; ab_conf.buf_depth=BLCK_BUF_DEPTH_GPU; ab_conf.buf_branch=BLCK_BUF_BRANCH_GPU;
 err_code=free_buf_entry(ab_conf,abg_occ[gpu_num],abg_occ_size[gpu_num],&blck_sizes_gpu[gpu_num][0],entry_num);
// if(err_code == 0 && DEBUG != 0) printf("\n#DEBUG(c_proc_bufs.cu:free_buf_entry_gpu): Entry deallocated: %d %d\n",gpu_num,entry_num); //debug
 return err_code;
}

static int const_args_link_init(int gpu_beg, int gpu_end)
/** This function initializes the linked list const_args_link[]
for GPU constant memory buffers (for each GPU in the range [gpu_beg..gpu_end]) **/
{
 if(gpu_beg >= 0 && gpu_end >= gpu_beg){
  for(int gpu_num=gpu_beg;gpu_num<=gpu_end;gpu_num++){
   if(gpu_num < MAX_GPUS_PER_NODE){
    const_args_ffe[gpu_num]=0; //first free entry for each GPU
    for(int i=0;i<MAX_GPU_ARGS;i++) const_args_link[gpu_num][i]=i+1; //linked list of free entries for each GPU
   }else{
    return 1;
   }
  }
 }
 return 0;
}

int const_args_entry_get(int gpu_num, int *entry_num)
/** This function returns the number of a free const_args[] entry for GPU#gpu_num **/
{
 if(const_args_ffe[gpu_num] >= 0 && const_args_ffe[gpu_num] < MAX_GPU_ARGS){ //free entry exists
  *entry_num=const_args_ffe[gpu_num];
   const_args_ffe[gpu_num]=const_args_link[gpu_num][const_args_ffe[gpu_num]];
  return 0;
 }else{ //no free entry found
  return 1;
 }
}

int const_args_entry_free(int gpu_num, int entry_num)
/** This function frees an entry of const_args[] for GPU#gpu_num **/
{
 if(entry_num >= 0 && entry_num < MAX_GPU_ARGS){ //valid entry number
  if(const_args_ffe[gpu_num] < MAX_GPU_ARGS && const_args_ffe[gpu_num] >= 0){
   const_args_link[gpu_num][entry_num]=const_args_ffe[gpu_num];
  }
  const_args_ffe[gpu_num]=entry_num;
  return 0;
 }else{ //invalid entry number
  return 1;
 }
}

#ifndef NO_GPU
__host__ int host_mem_alloc_pin(void **host_ptr, size_t tsize){
 hipError_t err=hipHostAlloc(host_ptr,tsize,hipHostMallocPortable);
 if(err != hipSuccess) return 1;
 return 0;
}

__host__ int host_mem_free_pin(void *host_ptr){
 hipError_t err=hipHostFree(host_ptr);
 if(err != hipSuccess) return 1;
 return 0;
}

__host__ int host_mem_register(void *host_ptr, size_t tsize){
 hipError_t err=hipHostRegister(host_ptr,tsize,hipHostMallocPortable);
 if(err != hipSuccess) return 1;
 return 0;
}

__host__ int host_mem_unregister(void *host_ptr){
 hipError_t err=hipHostUnregister(host_ptr);
 if(err != hipSuccess) return 1;
 return 0;
}

__host__ int gpu_mem_alloc(void **dev_ptr, size_t tsize)
{
 hipError_t err=hipMalloc(dev_ptr,tsize); if(err != hipSuccess) return 1;
 return 0;
}

__host__ int gpu_mem_free(void *dev_ptr)
{
 hipError_t err=hipFree(dev_ptr); if(err != hipSuccess) return 1;
 return 0;
}
#endif
